#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "helper_functions.h"
#include <fstream>
using namespace std;

// defining variables here
#ifndef THREADS_PER_BLOCK
#define THREADS_PER_BLOCK   128
#endif


#ifndef NUMTRIALS
#define NUMTRIALS   2048
#endif
// 4 096 4K
// 8 192 8K
// 16 384  16k
// 32 768  32k
// 65 536  64k
// 131 072   128k
// 262 144   256k
// 524 288   512k
// 1 048 576  1M


// these two #defines are just to label things
// other than that, they do nothing:
#define IN
#define OUT


// global variables
// ranges for the random numbers:
const float GMIN =	20.0;	// ground distance in meters
const float GMAX =	30.0;	// ground distance in meters
const float HMIN =	10.0;	// cliff height in meters
const float HMAX =	20.0;	// cliff height in meters
const float DMIN  =	10.0;	// distance to castle in meters
const float DMAX  =	20.0;	// distance to castle in meters
const float VMIN  =	10.0;	// intial cnnonball velocity in meters / sec
const float VMAX  =	30.0;	// intial cnnonball velocity in meters / sec
const float THMIN = 	30.0;	// cannonball launch angle in degrees
const float THMAX =	70.0;	// cannonball launch angle in degrees

const float GRAVITY =	-9.8;	// acceleraion due to gravity in meters / sec^2
const float TOL = 5.0;		// tolerance in cannonball hitting the castle in meters
ofstream fp;

// function prototypes:
void CudaCheckError( );
//inline float Radians( float d );
float		Ranf( float, float );
int		Ranf( int, int );
void		TimeOfDaySeed( );
__device__ float Radians( float d );
__global__ void MonteCarlo( float *dvs, float *dths, float *dgs, float *dhs, float *dds, int *dhits );



int main( int argc, char* argv[ ] )
{
    TimeOfDaySeed( );
    fp.open("montecarlo.csv", ofstream::app);
    printf("ThreadsPerBlock: %d  NUMTRIALS: %d  ", THREADS_PER_BLOCK, NUMTRIALS);
    int dev = findCudaDevice(argc, (const char **)argv);

    // better to define these here so that the rand() calls don't get into the thread timing:
    float *hvs   = new float [NUMTRIALS];
    float *hths  = new float [NUMTRIALS];
    float *hgs   = new float [NUMTRIALS];
    float *hhs   = new float [NUMTRIALS];
    float *hds   = new float [NUMTRIALS];
    int   *hhits = new int   [NUMTRIALS];

    // fill the random-value arrays:
    for( int n = 0; n < NUMTRIALS; n++ )
    {
        hvs[n]  = Ranf(  VMIN,  VMAX );
        hths[n] = Ranf( THMIN, THMAX );
        hgs[n]  = Ranf(  GMIN,  GMAX );
        hhs[n]  = Ranf(  HMIN,  HMAX );
        hds[n]  = Ranf(  DMIN,  DMAX );
        hhits[n] = 0;
    }


    // allocate device memory:
    float *dvs, *dths, *dgs, *dhs, *dds;
    int   *dhits;

    hipMalloc( &dvs,   NUMTRIALS*sizeof(float) );
    hipMalloc( &dths,  NUMTRIALS*sizeof(float) );
    hipMalloc( &dgs,   NUMTRIALS*sizeof(float) );
    hipMalloc( &dhs,   NUMTRIALS*sizeof(float) );
    hipMalloc( &dds,   NUMTRIALS*sizeof(float) );
    hipMalloc( &dhits, NUMTRIALS*sizeof(int) );
    CudaCheckError( );

    // copy host memory to the device:
    hipMemcpy( dvs,  hvs,  NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( dths, hths, NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( dgs,  hgs,  NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( dhs,  hhs,  NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( dds,  hds,  NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
    CudaCheckError( );

    // setup the execution parameters:
    dim3 grid( NUMTRIALS/THREADS_PER_BLOCK, 1, 1 );
    dim3 threads( THREADS_PER_BLOCK, 1, 1 );

    // allocate cuda events that we'll use for timing:
    hipEvent_t start, stop;
    hipEventCreate( &start );
    hipEventCreate( &stop  );
    CudaCheckError( );

    // let the gpu go quiet:
    hipDeviceSynchronize( );

    // record the start event:
    hipEventRecord( start, NULL );
    CudaCheckError( );

    // execute the kernel:
    MonteCarlo<<< grid, threads >>>( IN dvs, IN dths, IN dgs, IN dhs, IN dds, OUT dhits );

    // record the stop event:
    hipEventRecord( stop, NULL );
    CudaCheckError( );

    // wait for the stop event to complete:
    hipDeviceSynchronize( );
    hipEventSynchronize( stop );
    CudaCheckError( );

    float msecTotal = 0.0f;
    hipEventElapsedTime( &msecTotal, start, stop );
    CudaCheckError( );

    // compute and print the performance
    float TrialsPerSecond = (float) (NUMTRIALS / (float)(msecTotal/1000)); //unit is trials/second
    float performance = (float) (TrialsPerSecond / (1000000.)); //unit is MegaTrials/second
    printf("Performance: %6.3f  ", performance);
    fp << performance << ",";
    fp.close();
    
    // copy result from the device to the host:
    hipMemcpy( hhits, dhits, NUMTRIALS*sizeof(int), hipMemcpyDeviceToHost );
    CudaCheckError( );

    // add up the hhits[ ] array: :
    int Total_hits = 0;
    for (int i =0; i<NUMTRIALS; i++)
    {
        Total_hits += hhits[i];
    }

    // compute and print the probability:
    float Hit_probability = (float)Total_hits/ (float)NUMTRIALS * 100.0;
	printf("Probability: %%%6.3f\n", Hit_probability); //print newline character here
    

    // clean up host memory:
    delete [ ] hvs;
    delete [ ] hths;
    delete [ ] hgs;
    delete [ ] hhs;
    delete [ ] hds;
    delete [ ] hhits;

    // clean up device memory:
    hipFree( dvs );
    hipFree( dths );
    hipFree( dgs );
    hipFree( dhs );
    hipFree( dds );
    hipFree( dhits );
    CudaCheckError( );

	return 0;
}


void CudaCheckError( )
{
    hipError_t e = hipGetLastError( );
    if( e != hipSuccess )
    {
            fprintf( stderr, "CUDA failure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e) );
    }
}


// degrees-to-radians -- callable from the device:
__device__ float Radians( float d )
{
        return (M_PI/180.f) * d;
}


// the kernel:
__global__ void MonteCarlo( float *dvs, float *dths, float *dgs, float *dhs, float *dds, int *dhits )
{
    unsigned int gid = blockIdx.x*blockDim.x + threadIdx.x;

    // randomize everything:
    float v   = dvs[gid];
    float thr = Radians( dths[gid] );
    float vx  = v * cos(thr);
    float vy  = v * sin(thr);
    float  g  =  dgs[gid];
    float  h  =  dhs[gid];
    float  d  =  dds[gid];

    int numHits = 0;

    // see if the ball doesn't even reach the cliff:
    float t = -vy / ( 0.5*GRAVITY );
    float x = vx * t;
    if( x > g )
    {
        t = (float) (g/vx);
        float y = (float) vy*t + 0.5 * GRAVITY * t * t;
        if (y > h)
        {
            float a = 0.5*GRAVITY;
            float b = vy;
            float c = -h;
            float disc = b*b - 4.f*a*c;
            if(disc >= 0.)
            {
                disc = sqrtf( disc );
                float t1 = (-b + disc ) / ( 2.f*a );	// time to intersect high ground
                float t2 = (-b - disc ) / ( 2.f*a );	// time to intersect high ground

                // only care about the second intersection
                float tmax = t1;
                if( t2 > t1 )
                    tmax = t2;
                float upperDist = vx * tmax  -  g;
                if (fabs( upperDist - d ) <= TOL)
                {
                    numHits = 1;
                }
            }
        }       
    }
    dhits[gid] = numHits;
}


float Ranf( float low, float high )
{
        float r = (float) rand();               // 0 - RAND_MAX
        float t = r  /  (float) RAND_MAX;       // 0. - 1.

        return   low  +  t * ( high - low );
}

int Ranf( int ilow, int ihigh )
{
        float low = (float)ilow;
        float high = ceil( (float)ihigh );

        return (int) Ranf(low,high);
}


// call this if you want to force your program to use
// a different random number sequence every time you run it:
void TimeOfDaySeed( )
{
	struct tm y2k = { 0 };
	y2k.tm_hour = 0;   y2k.tm_min = 0; y2k.tm_sec = 0;
	y2k.tm_year = 100; y2k.tm_mon = 0; y2k.tm_mday = 1;

	time_t  timer;
	time( &timer );
	double seconds = difftime( timer, mktime(&y2k) );
	unsigned int seed = (unsigned int)( 1000.*seconds );    // milliseconds
	srand( seed );
}